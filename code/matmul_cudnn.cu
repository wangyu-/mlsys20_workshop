/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cuda_helper.h"

void Matmul::map(void)
{
  // create descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  int inputX = inputs[0].dim[0];
  int inputN = inputs[0].dim[1];
  int inputC = inputs[0].dim[2];
  int outputC = outputs[0].dim[2];
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputX, inputN, outputC, 1));
  if (actiMode != AC_MODE_NONE) {
    hipdnnActivationMode_t mode;
    switch (actiMode) {
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, mode,
        HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  }
  // allocate tensors
  size_t filterSize = sizeof(DATATYPE) * inputC * outputC;
  size_t outputSize = sizeof(DATATYPE) * inputX * inputN * outputC;
  checkCUDA(hipMalloc(&filterPtr, filterSize));
  checkCUDA(hipMalloc(&outputs[0].ptr, outputSize));
}

void Matmul::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
  if (actiMode != AC_MODE_NONE) {
    checkCUDNN(hipdnnDestroyActivationDescriptor(actiDesc));
  }
  checkCUDA(hipFree(filterPtr));
  checkCUDA(hipFree(outputs[0].ptr));
}

void Matmul::forward(void)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int inputX = inputs[0].dim[0];
  int inputN = inputs[0].dim[1];
  int batch = inputX * inputN;
  int inputC = inputs[0].dim[2];
  int outputC = outputs[0].dim[2];
  checkCUDA(hipblasSgemm(model->blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
      outputC, batch, inputC, &alpha, (float*)filterPtr, inputC,
      (float*)inputs[0].ptr, inputC, &beta, (float*)outputs[0].ptr, outputC));
  if (actiMode != AC_MODE_NONE)
    checkCUDNN(hipdnnActivationForward(model->dnn, actiDesc,
        &alpha, outputTensor, outputs[0].ptr,
        &beta, outputTensor, outputs[0].ptr));
}

void Model::measure_matmul_cost(Matmul* mm)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int inputX = mm->inputs[0].dim[0];
  int inputN = mm->inputs[0].dim[1];
  int batch = inputX * inputN;
  int inputC = mm->inputs[0].dim[2];
  int outputC = mm->outputs[0].dim[2];
  if (mm->actiMode != OpBase::AC_MODE_NONE) {
    hipdnnActivationMode_t mode;
    switch (mm->actiMode) {
      case OpBase::AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OpBase::AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OpBase::AC_MODE_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, mode,
        HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  }
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputX, inputN, outputC, 1));

  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
    checkCUDA(hipblasSgemm(blas, HIPBLAS_OP_T, HIPBLAS_OP_N,
        outputC, batch, inputC, &alpha, filterPtr, inputC,
        inputPtr, inputC, &beta, outputPtr, outputC));
    if (mm->actiMode != OpBase::AC_MODE_NONE)
      checkCUDNN(hipdnnActivationForward(dnn, actiDesc,
          &alpha, outputTensor, outputPtr,
          &beta, outputTensor, outputPtr));
  } 
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  mm->runtime = milliseconds / REPEAT_TIMES;
//#ifdef VERBOSE
  printf("measure[Matmul]: i(%d %d %d) o(%d) acti(%d) cost(%.4lf)\n",
         mm->inputs[0].dim[0], mm->inputs[0].dim[1], inputC, outputC,
         mm->actiMode, mm->runtime);
//#endif
}

