/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cuda_helper.h"

void Concat::map(void)
{
  size_t outputSize = sizeof(DATATYPE);
  for (int i = 0; i < outputs[0].numDim; i++)
    outputSize *= outputs[0].dim[i];
  checkCUDA(hipMalloc(&outputs[0].ptr, outputSize));
}

void Concat::unmap(void)
{
  checkCUDA(hipFree(outputs[0].ptr));
}

void Concat::forward(void)
{
  off_t offset = 0;
  for (int i = 0; i < numInputs; i++) {
    size_t size = sizeof(DATATYPE);
    for (int j = 0; j < inputs[i].numDim; j++)
      size *= inputs[i].dim[j];
    if (0&&needCopy[i])
      checkCUDA(hipMemcpyAsync(((char*)outputs[0].ptr) + offset,
                                inputs[i].ptr, size,
                                hipMemcpyDeviceToDevice));
    offset += size;
  }
}

void Model::measure_concat_cost(Concat* concat)
{
  string key=export_op_key(*concat);
  for (int j = 0; j < concat->numInputs; j++) {
	  if (concat->needCopy[j]) key+=",<1>";
	  else key+=",<0>";
	  }
  //printf("<pre_measure>, %s\n",key.c_str());
  if(mp.find(key)!=mp.end())
  {
	  concat->runtime=mp[key].runtime;
	  concat->power=mp[key].power;
          concat->energy=mp[key].power*mp[key].runtime;
	if(!mute)
	{
	  printf("<found from mp>, %s, ",key.c_str());
	  printf("runtime=%f power=%f energe=%f\n", mp[key].runtime, mp[key].power, mp[key].power*mp[key].runtime);
	}
        return ;

  }

  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
    for (int j = 0; j < concat->numInputs; j++) {
      if (0&&concat->needCopy[j]) {
        size_t size = sizeof(DATATYPE);
        for (int k = 0; k < concat->inputs[j].numDim; k++)
          size *= concat->inputs[j].dim[k];
        checkCUDA(hipMemcpyAsync(outputPtr, inputPtr, size,
                                  hipMemcpyDeviceToDevice));
      }
    }
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  //double runtime=concat->runtime = milliseconds / REPEAT_TIMES;


  long times=0;
  double current_time=get_current_time();
  double current_time2;
  start_check_power();
  for (int i = 0; ; i++,times++) {
    if(i%CHECK_TIME_PERIOD==0&&(current_time2=get_current_time())-current_time>measure_time) break;
    for (int j = 0; j < concat->numInputs; j++) {
      if (0&&concat->needCopy[j]) {
        size_t size = sizeof(DATATYPE);
        for (int k = 0; k < concat->inputs[j].numDim; k++)
          size *= concat->inputs[j].dim[k];
        checkCUDA(hipMemcpyAsync(outputPtr, inputPtr, size,
                                  hipMemcpyDeviceToDevice));
      }
    }
  }
  double power=finish_check_power();
  double runtime=concat->runtime = (current_time2-current_time)/times;
   
  printf("<measure>, %s, ",key.c_str());
  printf("runtime=%f power=%f energy=%f\n",runtime,power,power*runtime);
  concat->power=power;
  concat->energy=power*runtime;
  mp[key].runtime=runtime;
  mp[key].power=power;
  db_output<<key<<"|"<<runtime<<"|"<<power<<endl;
  db_output.flush();


#ifdef VERBOSE
  printf("measure[Concat]: cost(%.4lf)\n", concat->runtime);
#endif
}

