/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cuda_helper.h"

void Pool2D::map(void)
{
  // create descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
  int inputC = inputs[0].dim[1];
  int inputH = inputs[0].dim[2];
  int inputW = inputs[0].dim[3];
  // set descriptors
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, BATCH_SIZE, inputC, inputH, inputW));
  hipdnnPoolingMode_t mode;
  if (type == OP_POOL2D_MAX)
    mode = HIPDNN_POOLING_MAX;
  else if (type == OP_POOL2D_AVG)
    mode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
  checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc, mode, HIPDNN_PROPAGATE_NAN,
      kernelH, kernelW, padH, padW, strideH, strideW));
  int n, c, h, w;
  checkCUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc, 
      inputTensor, &n, &c, &h, &w));
  assert(n == BATCH_SIZE);
  assert(c == inputC);
  assert(outputs[0].dim[2] == h);
  assert(outputs[0].dim[3] == w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  if (relu) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, HIPDNN_ACTIVATION_RELU,
        HIPDNN_PROPAGATE_NAN, 0.0));
  }
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE) * n * c * h * w;
  checkCUDA(hipMalloc(&outputs[0].ptr, outputSize));
}

void Pool2D::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
  checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
  if (relu) {
    checkCUDNN(hipdnnDestroyActivationDescriptor(actiDesc));
  }
  // free tensors
  checkCUDA(hipFree(outputs[0].ptr));
}

void Pool2D::forward(void)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  checkCUDNN(hipdnnPoolingForward(model->dnn, poolDesc,
      &alpha, inputTensor, inputs[0].ptr,
      &beta, outputTensor, outputs[0].ptr));
  if (relu) {
    checkCUDNN(hipdnnActivationForward(model->dnn, actiDesc,
        &alpha, outputTensor, outputs[0].ptr,
        &beta, outputTensor, outputs[0].ptr));
  }
}

void Model::measure_pool2d_cost(Pool2D* pool)
{
  string key=export_op_key(*pool);
  //printf("<pre_measure>, %s\n",key.c_str());
  if(mp.find(key)!=mp.end())
  {
	  pool->runtime=mp[key].runtime;
	  pool->power=mp[key].power;
          pool->energy=mp[key].power*mp[key].runtime;
	if(!mute)
	{
	  printf("<found from mp>, %s, ",key.c_str());
	  printf("runtime=%f power=%f energe=%f\n", mp[key].runtime, mp[key].power, mp[key].power*mp[key].runtime);
	}
	return ;
     
  }
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int inputC = pool->inputs[0].dim[1];
  int inputH = pool->inputs[0].dim[2];
  int inputW = pool->inputs[0].dim[3];
  int outputH = pool->outputs[0].dim[2];
  int outputW = pool->outputs[0].dim[3];
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, BATCH_SIZE, inputC, inputH, inputW));
  hipdnnPoolingMode_t mode;
  if (pool->type == OpBase::OP_POOL2D_MAX)
    mode = HIPDNN_POOLING_MAX;
  else if (pool->type == OpBase::OP_POOL2D_AVG)
    mode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
  checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc, mode,
      HIPDNN_PROPAGATE_NAN, pool->kernelH, pool->kernelW, pool->padH, pool->padW,
      pool->strideH, pool->strideW));
  checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  int n, c, h, w;
  checkCUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc,
      inputTensor, &n, &c, &h, &w));
  assert(n == BATCH_SIZE);
  assert(c == inputC);
  assert(outputH == h);
  assert(outputW == w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  size_t inputSize = sizeof(DATATYPE) * BATCH_SIZE * inputC * inputH * inputW;
  size_t outputSize = sizeof(DATATYPE) * BATCH_SIZE * inputC * outputH * outputW;
  assert(inputSize < MAX_TENSOR_SIZE);
  assert(outputSize < MAX_TENSOR_SIZE);

  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
    checkCUDNN(hipdnnPoolingForward(dnn, poolDesc,
        &alpha, inputTensor, inputPtr,
        &beta, outputTensor, outputPtr));
    if (pool->relu) {
      checkCUDNN(hipdnnActivationForward(dnn, actiDesc,
          &alpha, outputTensor, outputPtr,
          &beta, outputTensor, outputPtr));
    }
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  //double runtime=pool->runtime = milliseconds / REPEAT_TIMES;

  {
	  long times=0;
	  double current_time=get_current_time();
	  for (int i = 0; ; i++,times++) {
		  if(i%CHECK_TIME_PERIOD==0&&get_current_time()-current_time>stress_time) break;
		  checkCUDNN(hipdnnPoolingForward(dnn, poolDesc,
					  &alpha, inputTensor, inputPtr,
					  &beta, outputTensor, outputPtr +OLD_SIZE*(i%MOD) ));
		  if (pool->relu) {
			  checkCUDNN(hipdnnActivationForward(dnn, actiDesc,
						  &alpha, outputTensor, outputPtr,
						  &beta, outputTensor, outputPtr));
		  }
	  }
	  checkCUDA(hipDeviceSynchronize());
  }

  sleep(idle_time);

  long times=0;
  double current_time=get_current_time();
  start_check_power();
  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; ; i++,times++) {
    if(i%CHECK_TIME_PERIOD==0&&get_current_time()-current_time>measure_time) break;
    checkCUDNN(hipdnnPoolingForward(dnn, poolDesc,
        &alpha, inputTensor, inputPtr,
        &beta, outputTensor, outputPtr +OLD_SIZE*(i%MOD) ));
    if (pool->relu) {
      checkCUDNN(hipdnnActivationForward(dnn, actiDesc,
          &alpha, outputTensor, outputPtr,
          &beta, outputTensor, outputPtr));
    }
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float gpu_time;
  hipEventElapsedTime(&gpu_time, startEvent, endEvent);

  double power=finish_check_power();
  double runtime=pool->runtime = (gpu_time)/times;

  printf("<measure>, %s, ",key.c_str());
  printf("runtime=%f power=%f energy=%f\n",runtime,power,power*runtime);
  pool->power=power;
  pool->energy=power*runtime;
  mp[key].runtime=runtime;
  mp[key].power=power;
  db_output<<key<<"|"<<runtime<<"|"<<power<<endl;
  db_output.flush();
#ifdef VERBOSE
  printf("measure[Pool2D]: i(%d %d %d %d) k(%d %d) s(%d %d) p(%d %d) cost(%.4lf)\n",
         BATCH_SIZE, inputC, inputH, inputW, pool->kernelH, pool->kernelW,
         pool->strideH, pool->strideW, pool->padH, pool->padW, pool->runtime);
#endif
}

