/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cuda_helper.h"

void Conv2D::map(void)
{
  // create descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  int inputC = inputs[0].dim[1];
  int inputH = inputs[0].dim[2];
  int inputW = inputs[0].dim[3];
  // set descriptors
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, BATCH_SIZE, inputC, inputH, inputW));
  checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, outputC, 1, 1));
  checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT,
      HIPDNN_TENSOR_NCHW, outputC, inputC, kernelH, kernelW));
  //checkCUDNN(hipdnnSetConvolutionMathType(convDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
  //checkCUDNN(hipdnnSetConvolutionMathType(convDesc, HIPDNN_DEFAULT_MATH));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padH, padW,
      strideH, strideW, 1/*dilationH*/, 1/*dilationW*/,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc,
      inputTensor, filterDesc, &n, &c, &h, &w));
  assert(n == BATCH_SIZE);
  assert(c == outputC);
  assert(outputs[0].dim[2] == h);
  assert(outputs[0].dim[3] == w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  if (relu) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  }
  int outputC = c;
  int outputH = h;
  int outputW = w;
  // allocate tensors
  size_t filterSize = sizeof(DATATYPE) * inputC * outputC * kernelH * kernelW;
  size_t outputSize = sizeof(DATATYPE) * BATCH_SIZE * outputC * outputH * outputW;
  size_t biasSize = sizeof(DATATYPE) * outputC;
  checkCUDA(hipMalloc(&filterPtr, filterSize));
  checkCUDA(hipMalloc(&biasPtr, biasSize));
  checkCUDA(hipMalloc(&outputs[0].ptr, outputSize));
}

void Conv2D::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
  checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
  checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
  if (relu) {
    checkCUDNN(hipdnnDestroyActivationDescriptor(actiDesc));
  }
  // free tensors
  checkCUDA(hipFree(filterPtr));
  checkCUDA(hipFree(outputs[0].ptr));
  checkCUDA(hipFree(biasPtr));
}

void Conv2D::forward(void)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  if (relu) {
    checkCUDNN(cudnnConvolutionBiasActivationForward(
        model->dnn, &alpha, inputTensor, inputs[0].ptr, filterDesc, filterPtr,
        convDesc, fwdAlgo, model->workSpace, model->workSpaceSize,
        &beta, outputTensor, outputs[0].ptr, biasTensor, biasPtr, actiDesc,
        outputTensor, outputs[0].ptr));
  } else {
    checkCUDNN(hipdnnConvolutionForward(
        model->dnn, &alpha, inputTensor, inputs[0].ptr, filterDesc, filterPtr,
        convDesc, fwdAlgo, model->workSpace, model->workSpaceSize,
        &beta, outputTensor, outputs[0].ptr));
    checkCUDNN(hipdnnAddTensor(model->dnn, &alpha, biasTensor, biasPtr,
        &alpha, outputTensor, outputs[0].ptr));
  }
}

void Model::measure_conv2d_cost(Conv2D* conv)
{
  assert(conv->algo_cost_mp.empty());
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int inputC = conv->inputs[0].dim[1];
  int inputH = conv->inputs[0].dim[2];
  int inputW = conv->inputs[0].dim[3];
  int outputC = conv->outputs[0].dim[1];
  int outputH = conv->outputs[0].dim[2];
  int outputW = conv->outputs[0].dim[3];
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, BATCH_SIZE, inputC, inputH, inputW));
  checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, outputC, 1, 1));
  checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT,
      HIPDNN_TENSOR_NCHW, outputC, inputC, conv->kernelH, conv->kernelW));
  //checkCUDNN(hipdnnSetConvolutionMathType(convDesc, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, conv->padH, conv->padW,
      conv->strideH, conv->strideW, 1/*dilationH*/, 1/*dilationW*/,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc,
      inputTensor, filterDesc, &n, &c, &h, &w));
  assert(n == BATCH_SIZE);
  assert(c == outputC);
  assert(outputH == h);
  assert(outputW == w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  size_t inputSize = sizeof(DATATYPE) * BATCH_SIZE * inputC * inputH * inputW;
  size_t filterSize = sizeof(DATATYPE) * inputC * outputC
                      * conv->kernelH * conv->kernelW;
  size_t outputSize = sizeof(DATATYPE) * BATCH_SIZE * outputC * outputH * outputW;
  assert(inputSize < MAX_TENSOR_SIZE);
  assert(filterSize < MAX_TENSOR_SIZE);
  assert(outputSize < MAX_TENSOR_SIZE);

  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
      dnn, inputTensor, inputPtr, filterDesc, filterPtr, convDesc,
      outputTensor, outputPtr, reqAlgCnt, &cnt, perfResults,
      workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
#ifdef VERBOSE
  for (int i = 0; i < cnt; i++) {
    printf("fwdAlgo(%d) time(%.2lfms) space(%zuMB)\n", perfResults[i].algo,
           perfResults[i].time, perfResults[i].memory / 1024 / 1024);
  }
#endif

  double best_cost=999999999999999.0;
  int first=1;
  int begin=cnt-1;
  if(use_perf_order)
  {
     begin=0;
  }

  for(int idx=begin;idx>=0;idx--)
{
  if(int(perfResults[idx].status)!=0) continue;
  hipdnnConvolutionFwdAlgo_t current_algo=perfResults[idx].algo;
  //printf("<<<%d>>>\n",int(perfResults[idx].status));
  //conv->fwdAlgo = (hipdnnConvolutionFwdAlgo_t)2;
 
  string key=export_op_key(*conv)+",<"+to_string(current_algo)+">";
  double runtime;
  double power;
  double energy;
  //printf("<pre_measure>, %s\n",key.c_str());

  if(mp.find(key)!=mp.end())
  {
	  runtime=mp[key].runtime;
	  power=mp[key].power;
          energy=mp[key].power*mp[key].runtime;
	  
	  if(!mute)
	  {
		  printf("<found from mp>, %s, ",key.c_str());
		  printf("runtime=%f power=%f energe=%f\n", mp[key].runtime, mp[key].power, mp[key].power*mp[key].runtime);
	  }
	  goto end;

  }
  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
    if (conv->relu) {
      checkCUDNN(cudnnConvolutionBiasActivationForward(
          dnn, &alpha, inputTensor, inputPtr, filterDesc, filterPtr,
          convDesc, current_algo, workSpace, workSpaceSize,
          &beta, outputTensor, outputPtr, biasTensor, biasPtr, actiDesc,
          outputTensor, outputPtr));
    } else {
      checkCUDNN(hipdnnConvolutionForward(
          dnn, &alpha, inputTensor, inputPtr, filterDesc, filterPtr,
          convDesc, current_algo, workSpace, workSpaceSize,
          &beta, outputTensor, outputPtr));
      checkCUDNN(hipdnnAddTensor(dnn, &alpha, biasTensor, biasPtr,
          &alpha, outputTensor, outputPtr));
    }
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  //double runtime=conv->runtime = milliseconds / REPEAT_TIMES;
  
  {
  long times=0;
  double current_time=get_current_time();
  double current_time2;
  start_check_power();
  for (int i = 0; ; i++,times++) {
    if(i%CHECK_TIME_PERIOD==0&&(current_time2=get_current_time())-current_time>measure_time) break;
    if (conv->relu) {
      checkCUDNN(cudnnConvolutionBiasActivationForward(
          dnn, &alpha, inputTensor, inputPtr, filterDesc, filterPtr,
          convDesc, current_algo, workSpace, workSpaceSize,
          &beta, outputTensor, outputPtr, biasTensor, biasPtr, actiDesc,
          outputTensor, outputPtr));
    } else {
      checkCUDNN(hipdnnConvolutionForward(
          dnn, &alpha, inputTensor, inputPtr, filterDesc, filterPtr,
          convDesc, current_algo, workSpace, workSpaceSize,
          &beta, outputTensor, outputPtr));
      checkCUDNN(hipdnnAddTensor(dnn, &alpha, biasTensor, biasPtr,
          &alpha, outputTensor, outputPtr));
    }
  }
  power=finish_check_power();
  runtime= (current_time2-current_time)/times;
  energy=power*runtime;

  printf("<measure>, %s, ",key.c_str());
  printf("runtime=%f power=%f energy=%f\n",runtime,power,power*runtime);

  mp[key].runtime=runtime;
  mp[key].power=power;
  db_output<<key<<"|"<<runtime<<"|"<<power<<endl;
  db_output.flush();
#ifdef VERBOSE
  printf("measure[Conv2D]: i(%d %d %d %d) o(%d) k(%d %d) s(%d %d) p(%d %d) cost(%.4lf)\n",
         BATCH_SIZE, inputC, inputH, inputW, outputC, conv->kernelH, conv->kernelW,
         conv->strideH, conv->strideW, conv->padH, conv->padW, conv->runtime);
#endif
  }
  end:
  cost_t tmp;
  tmp.runtime=runtime;
  tmp.energy=energy;
  conv->algo_cost_mp[current_algo]=tmp;

  double cost= cost_func(runtime,power);
  if(first||cost<best_cost||use_perf_order)
  {
	  first=0;
	  best_cost=cost;
	  conv->runtime=runtime;
	  conv->power=power;
	  conv->energy=energy;
	  conv->fwdAlgo=current_algo;
  }
}
}

