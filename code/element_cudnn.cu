#include "hip/hip_runtime.h"
/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cuda_helper.h"

void Element::map(void)
{
  // create descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  // set descriptors
  int inputN = inputs[0].dim[0];
  int inputC = max(inputs[0].dim[1], 1);
  int inputH = max(inputs[0].dim[2], 1);
  int inputW = max(inputs[0].dim[3], 1);
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));

  hipdnnOpTensorOp_t opType;
  switch (type) {
    case OP_EW_ADD:
      opType = HIPDNN_OP_TENSOR_ADD;
      break;
    case OP_EW_MUL:
      opType = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(opDesc, opType, HIPDNN_DATA_FLOAT,
      HIPDNN_NOT_PROPAGATE_NAN));
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE);
  for (int i = 0; i < outputs[0].numDim; i++)
    outputSize *= outputs[0].dim[i];
  checkCUDA(hipMalloc(&outputs[0].ptr, outputSize));
}

void Element::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
  checkCUDNN(hipdnnDestroyOpTensorDescriptor(opDesc));
  checkCUDA(hipFree(outputs[0].ptr));
}

void Element::forward(void)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  checkCUDNN(hipdnnOpTensor(model->dnn, opDesc, &alpha, inputTensor, inputs[0].ptr,
      &alpha, inputTensor, inputs[1].ptr, &beta, inputTensor, outputs[0].ptr));
}

void Model::measure_element_cost(Element* ele)
{
  string key=export_op_key(*ele);
  //printf("<pre_measure>, %s\n",key.c_str());

  if(mp.find(key)!=mp.end())
  {
	  ele->runtime=mp[key].runtime;
	  ele->power=mp[key].power;
          ele->energy=mp[key].power*mp[key].runtime;
	if(!mute)
	{
	  printf("<found from mp>, %s, ",key.c_str());
	  printf("runtime=%f power=%f energe=%f\n", mp[key].runtime, mp[key].power, mp[key].power*mp[key].runtime);
	}
        return ;

  }

  
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int inputN = ele->inputs[0].dim[0];
  int inputC = max(ele->inputs[0].dim[1], 1);
  int inputH = max(ele->inputs[0].dim[2], 1);
  int inputW = max(ele->inputs[0].dim[3], 1);
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));

  hipdnnOpTensorOp_t opType;
  switch (ele->type) {
    case OpBase::OP_EW_ADD:
      opType = HIPDNN_OP_TENSOR_ADD;
      break;
    case OpBase::OP_EW_MUL:
      opType = HIPDNN_OP_TENSOR_MUL;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetOpTensorDescriptor(opDesc, opType, HIPDNN_DATA_FLOAT,
      HIPDNN_NOT_PROPAGATE_NAN));

  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
    checkCUDNN(hipdnnOpTensor(dnn, opDesc, &alpha, inputTensor, inputPtr,
        &alpha, inputTensor, filterPtr, &beta, inputTensor, outputPtr));
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
//  double runtime=ele->runtime = milliseconds / REPEAT_TIMES;
  {
	  long times=0;
	  double current_time=get_current_time();
	  for (int i = 0; ; i++,times++) {
		  if(i%CHECK_TIME_PERIOD==0&&get_current_time()-current_time>stress_time) break;
		  checkCUDNN(hipdnnOpTensor(dnn, opDesc, &alpha, inputTensor, inputPtr,
					  &alpha, inputTensor, filterPtr, &beta, inputTensor, outputPtr));
	  }
	  checkCUDA(hipDeviceSynchronize());
  }

  sleep(idle_time);

  long times=0; 
  double current_time=get_current_time();
  start_check_power();
  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));

  for (int i = 0; ; i++,times++) {
    if(i%CHECK_TIME_PERIOD==0&&get_current_time()-current_time>measure_time) break;
    checkCUDNN(hipdnnOpTensor(dnn, opDesc, &alpha, inputTensor, inputPtr,
        &alpha, inputTensor, filterPtr, &beta, inputTensor, outputPtr));
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float gpu_time;
  hipEventElapsedTime(&gpu_time, startEvent, endEvent);

  double power=finish_check_power();
  double runtime=ele->runtime = gpu_time/times;

  printf("<measure>, %s, ",key.c_str());
  printf("runtime=%f power=%f energy=%f\n",runtime,power,power*runtime);
  ele->power=power;
  ele->energy=power*runtime;
  mp[key].runtime=runtime;
  mp[key].power=power;
  db_output<<key<<"|"<<runtime<<"|"<<power<<endl;
  db_output.flush();
#ifdef VERBOSE
  printf("measure[Element]: i(%d %d %d %d) type(%d) cost(%.4lf)\n",
         ele->inputs[0].dim[0], ele->inputs[0].dim[1], ele->inputs[0].dim[2],
         ele->inputs[0].dim[3], ele->type, ele->runtime);
#endif
}

