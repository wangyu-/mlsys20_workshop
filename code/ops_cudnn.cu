#include "hip/hip_runtime.h"
/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ops.h"
#include "cuda_helper.h"

Model::Model(bool training)
: isTraining(training)
{
  //checkCUDA(hipSetDevice(0));
  checkCUDNN(hipdnnCreate(&dnn));
  checkCUDA(hipblasCreate(&blas));
  workSpaceSize = WORK_SPACE_SIZE;
  global_unique_id = 100;
  checkCUDA(hipMalloc(&workSpace, workSpaceSize));
  //printf("handle.workSpace = 0x%p\n", workSpace);
  // create all descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&scaleTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  // allocate tensors for measuring performance
  checkCUDA(hipMalloc(&inputPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&biasPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&outputPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&filterPtr, MAX_TENSOR_SIZE));
  // create tensors for batch norm
  checkCUDA(hipMalloc(&scalePtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&runningMean, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&runningVar, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&saveMean, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&saveVar, MAX_TENSOR_SIZE));
  // create cuda events
  checkCUDA(hipEventCreate(&startEvent));
  checkCUDA(hipEventCreate(&endEvent));
}

float Model::measure_oplist_runtime(const std::vector<OpBase*>& opBaseList)
{
  const int num_runs = 100;
  // warmup
  for (int i = 0; i < opBaseList.size(); i++)
    opBaseList[i]->forward();
  // measure runtime
  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int times = 0; times < num_runs; times++) {
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  double runtime=milliseconds/num_runs;

  double power_time=measure_time/ (runtime);
  
  start_check_power();
  double t=get_current_time();
  for (int times = 0; times < power_time; times++) {
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();
    checkCUDA(hipDeviceSynchronize());
  }
  t=get_current_time()-t;
  double power=finish_check_power();
  double new_run_time=t/power_time;
  power=power_no_idle(power);
  printf("\n    Measured power=%f energy=%f\n",power,new_run_time*power);
  
  return new_run_time;
}

void* Model::allocate_memory(size_t size)
{
  void* ptr;
  checkCUDA(hipMalloc(&ptr, size));
  return ptr;
}
